#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void addKernel(int *a, int *b, int *c, int size)
{
    int tid = ((blockDim.x * threadIdx.y) + threadIdx.x) + ((blockDim.x * blockDim.y) * threadIdx.z);


    int blockId = blockIdx.x + (gridDim.x * blockIdx.y) + (gridDim.x * gridDim.y * blockIdx.z);

    int num_threads_in_block = blockId * blockDim.x * blockDim.y;

    int block_offset = num_threads_in_block * blockDim.z;
    int gid = tid + block_offset;

    if (gid < size)
    {
        c[gid] = a[gid] + b[gid];
    }

}

int main()
{
    int size = 10000;
    
    int NO_BYTES = size * sizeof(int);

    int* h_a, * h_b, * gpu_results;

    h_a = (int*)malloc(NO_BYTES);

    time_t t;

    srand((unsigned)time(&t));
    for (int i = 0; i < size; i++)
    {
        h_a[i] = (int)(rand() & 0xFF);
    }
    for (int i = 0; i < size; i++)
    {
        h_b[i] = (int)(rand() & 0xFF);
    }

    int* d_a, * d_b, * d_c;
    hipMalloc((int**)&d_a, NO_BYTES);
    hipMalloc((int**)&d_b, NO_BYTES);
    hipMalloc((int**)&d_c, NO_BYTES);

    hipMemcpy(d_a, h_a, NO_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, NO_BYTES, hipMemcpyHostToDevice);

    dim3 block(5, 5, 5);
    dim3 grid(5, 5, 5);

    addKernel << <grid, block >> > (d_a, d_b, d_c, size);
    hipDeviceSynchronize();


}
